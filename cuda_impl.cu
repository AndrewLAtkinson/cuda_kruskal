#include "hip/hip_runtime.h"
#include<time.h>
#include<stdlib.h>
#include<stdio.h>

#define TPB 1

void TopDownMergeSort(int *A[], int n, int *B[]);
void TopDownSplitMerge(int *A[], int iBegin, int iEnd, int *B[]);
void TopDownMerge(int *A[], int iBegin, int iMiddle, int iEnd, int *B[]);
void CopyArray(int *B[], int iBegin, int iEnd, int *A[]);

__global__ void kruskal( int *a, double *h, double *area, double *y, double *n){

		int i = threadIdx.y + blockIdx.y * blockDim.y;
		int j = threadIdx.x + blockIdx.x * blockDim.x;

		if( i < *grid && j < *grid){
			y[index] = (*a) + index * (*h);
			area[index] = 1 + y[index] + sin(2*y[index]);
		}
}

int main( int argc, int **argv){

	clock_t begin, end;
	double time_spent;

	begin = clock();

	int tree[grid][grid], mst[grid][grid], nodeSet[grid], r, count = 0;
	srand(time(NULL));

	for(int i = 0; i < grid; i++){
		for(int j = 0; j < grid; j++){
			tree[i][j] = 0;
			mst[i][j] = 0;
		}
	}

	for(int i = 0; i < grid; i++){
		for(int j = 0; j < grid; j++){
			r = rand() % 100;
			if(!tree[i][j] && i < j && r){
				tree[j][i] = r;
				tree[i][j] = r;
				count++;
			}
		}
	}

	int **minedge = malloc(sizeof(int*)*count);
	int **tempedge = malloc(sizeof(int*)*count);

	for( int i = 0; i < count; i++){
		minedge[i] = malloc(2);
		tempedge[i] = malloc(2);
	}

	count = 0;

	for(int i = 0; i < grid; i++){
		for(int j = 0; j < grid; j++){
			if(tree[i][j] > 0 && i < j){
				minedge[count][0] = tree[i][j];
				minedge[count][1] = (i*grid)+j;
				count++;
			}
		}
	}

	for(int i = 0; i < grid; i++)
		nodeSet[i] = i;

	TopDownMergeSort( minedge, count, tempedge);

	int nodeA, nodeB;

	for(int i = 0; i < count; i++){
		nodeA = minedge[i][1]%grid;
		nodeB = minedge[i][1]/grid;
		if(nodeSet[nodeA] != nodeSet[nodeB]){
			for(int j = 0; j < grid; j++){
				if(nodeSet[j] == nodeSet[nodeB] && j != nodeB)
					nodeSet[j] = nodeSet[nodeA];
			}
			nodeSet[nodeB] = nodeSet[nodeA];
			mst[nodeA][nodeB] = minedge[i][0];
			mst[nodeB][nodeA] = minedge[i][0];
		}
	}

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

	for(int i = -1; i < grid; i++){
		if( i >= 0)
			printf("\n%d ", i);
		else
			printf("\n  ");
		for(int j = 0; j < grid; j++){
			if(i == -1)
				printf("%d ", j);
			else
				printf("%c%c ", j <= i ? ' ' : tree[i][j]/10 > 0 ? tree[i][j]/10 + '0' : '\0', j <= i ? '\0' : tree[i][j]%10 + '0');
		}
		printf("\n");
	}

	for(int i = -1; i < grid; i++){
		if( i >= 0)
			printf("\n%d ", i);
		else
			printf("\n  ");
		for(int j = 0; j < grid; j++){
			if(i == -1)
				printf("%d ", j);
			else
				printf("%c%c ", j <= i ? ' ' : mst[i][j]/10 > 0 ? mst[i][j]/10 + '0' : '\0', j <= i ? '\0' : mst[i][j]%10 + '0');
		}
		printf("\n");
	}

	printf("timespent = %lf\n", time_spent);
}

// Array A[] has the items to sort; array B[] is a work array.
void TopDownMergeSort(int *A[], int n, int *B[])
{
    TopDownSplitMerge(A, 0, n, B);
}

// iBegin is inclusive; iEnd is exclusive (A[iEnd] is not in the set).
void TopDownSplitMerge(int *A[], int iBegin, int iEnd, int *B[])
{
    if(iEnd - iBegin < 2)                       // if run size == 1
        return;                                 //   consider it sorted
    // recursively split runs into two halves until run size == 1,
    // then merge them and return back up the call chain
    int iMiddle = (iEnd + iBegin) / 2;              // iMiddle = mid point
    TopDownSplitMerge(A, iBegin,  iMiddle, B);  // split / merge left  half
    TopDownSplitMerge(A, iMiddle,    iEnd, B);  // split / merge right half
    TopDownMerge(A, iBegin, iMiddle, iEnd, B);  // merge the two half runs
    CopyArray(B, iBegin, iEnd, A);              // copy the merged runs back to A
}

//  Left half is A[iBegin :iMiddle-1].
// Right half is A[iMiddle:iEnd-1   ].
void TopDownMerge(int *A[], int iBegin, int iMiddle, int iEnd, int *B[])
{
    int i = iBegin, j = iMiddle;
    
    // While there are elements in the left or right runs...
    for (int k = iBegin; k < iEnd; k++) {
        // If left run head exists and is <= existing right run head.
        if (i < iMiddle && (j >= iEnd || A[i][0] <= A[j][0])) {
            B[k][0] = A[i][0];
			B[k][1] = A[i][1];
            i = i + 1;
        } else {
            B[k][0] = A[j][0];
            B[k][1] = A[j][1];
            j = j + 1;    
        }
    } 
}

void CopyArray(int *B[], int iBegin, int iEnd, int *A[])
{
    for(int k = iBegin; k < iEnd; k++){
		A[k][0] = B[k][0];
		A[k][1] = B[k][1];
	}
}

